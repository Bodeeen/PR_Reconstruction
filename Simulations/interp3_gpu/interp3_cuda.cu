
#include <hip/hip_runtime.h>
__global__ void interp3_cuda(
	float * vOutput,
	int     nPoints,
	int     xSize,
	int     ySize,
	int     zSize,
	float * gridX,
	float * gridY,
	float * gridZ,
	float * vInput,
	float * xInterp,
	float * yInterp,
	float * zInterp)
{
	int idx = blockDim.x * (gridDim.x * blockIdx.y + blockIdx.x) + threadIdx.x;
	if (idx >= nPoints)
	{
		return;
	}
	
	float x = xInterp[idx];
	float y = yInterp[idx];
	float z = zInterp[idx];

	if (x < gridX[0] || x > gridX[xSize-1] || 
		y < gridY[0] || y > gridY[ySize-1] || 
		z < gridZ[0] || z > gridZ[zSize-1])
	{
		vOutput[idx] = 0.0f;
		return;
	}
	
	float x0, y0, z0, x1, y1, z1;
	int ibx, itx, iby, ity, ibz, itz, im;

	ibx = 0;
	itx = xSize - 1;
	while (ibx < (itx-1))
	{
		im = ((ibx + itx) >> 1);
		if (x <= gridX[im])
		{
			itx = im;
		}		
		else
		{
			ibx = im;
		}
	}
	x0 = gridX[ibx];
	x1 = gridX[itx];
		
	iby = 0;
	ity = ySize - 1;
	while (iby < (ity-1))
	{
		im = ((iby + ity) >> 1);
		if (y <= gridY[im])
		{
			ity = im;
		}		
		else
		{
			iby = im;
		}
	}
	y0 = gridY[iby];
	y1 = gridY[ity];

	ibz = 0;
	itz = zSize - 1;
	while (ibz < (itz-1))
	{
		im = ((ibz + itz) >> 1);
		if (z <= gridZ[im])
		{
			itz = im;
		}
		else
		{
			ibz = im;
		}
	}
	z0 = gridZ[ibz];
	z1 = gridZ[itz];

	int sliceDim = xSize * ySize;
	int zOff0 = sliceDim * ibz;
	int zOff1 = zOff0 + sliceDim;
	int yOff0 = ySize * ibx;
	int yOff1 = yOff0 + ySize;

	float ax0 = (x - x0) / (x1 - x0);
	float ay0 = (y - y0) / (y1 - y0);
	float az0 = (z - z0) / (z1 - z0);
	float ax1 = 1.0f - ax0;
	float ay1 = 1.0f - ay0;
	
	float v000 = vInput[zOff0 + yOff0 + iby];
	float v001 = vInput[zOff0 + yOff0 + ity];
	float v010 = vInput[zOff0 + yOff1 + iby];
	float v011 = vInput[zOff0 + yOff1 + ity];
	float v100 = vInput[zOff1 + yOff0 + iby];
	float v101 = vInput[zOff1 + yOff0 + ity];
	float v110 = vInput[zOff1 + yOff1 + iby];
	float v111 = vInput[zOff1 + yOff1 + ity];
	
	float v00 = v000 * ay1 + v001 * ay0;
	float v01 = v010 * ay1 + v011 * ay0;
	float v10 = v100 * ay1 + v101 * ay0;
	float v11 = v110 * ay1 + v111 * ay0;
	
	float v0 = v00 * ax1 + v01 * ax0;
	float v1 = v10 * ax1 + v11 * ax0;

	vOutput[idx] = v0 * (1.0f - az0) + v1 * az0;
}